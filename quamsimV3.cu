#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sys/time.h>
#include <inttypes.h>
#include <math.h>
#include <hip/hip_runtime.h>

using namespace std;
uint64_t InputMatrixSize = 0;

__global__ void QuantumGate(float *A, float *B, float *C, uint64_t Alength, uint64_t Blength)
{
    uint64_t Aposition = 2 * threadIdx.x;
    uint64_t Bposition = blockIdx.x;
    float MatrixResult[4];
    __shared__ float SharedMemMatrix[64];

    if ((Aposition < Alength) && (Bposition < Blength))
    {
        if (Aposition == 0)
        {
            for (uint64_t Index = 0; Index < 64; Index++)
                SharedMemMatrix[Index] = B[(Bposition * 64) + Index];
        }
        __syncthreads();
        for (uint64_t QuBitOperationCount = 0; QuBitOperationCount < 6; QuBitOperationCount++)
        {
            uint64_t QbitPower = 1 << QuBitOperationCount; 
            uint64_t Remainder = Aposition % QbitPower;
            uint64_t Remainder1 = (Aposition + 1) % QbitPower;

            MatrixResult[0] = 0;
            MatrixResult[1] = 0;
            MatrixResult[2] = 0;
            MatrixResult[3] = 0;
            for (uint64_t Iteration = 0; Iteration < 2; Iteration++)
            {
                MatrixResult[0] += A[(QuBitOperationCount * 4) + Iteration + 0] * SharedMemMatrix[(Iteration * QbitPower) + ((Aposition - Remainder) * 2) + Remainder];
                MatrixResult[1] += A[(QuBitOperationCount * 4) + Iteration + 2] * SharedMemMatrix[(Iteration * QbitPower) + ((Aposition - Remainder) * 2) + Remainder];
                MatrixResult[2] += A[(QuBitOperationCount * 4) + Iteration + 0] * SharedMemMatrix[(Iteration * QbitPower) + ((Aposition + 1 - Remainder1) * 2) + Remainder1];
                MatrixResult[3] += A[(QuBitOperationCount * 4) + Iteration + 2] * SharedMemMatrix[(Iteration * QbitPower) + ((Aposition + 1 - Remainder1) * 2) + Remainder1];
            }
            SharedMemMatrix[(0 * QbitPower) + ((Aposition - Remainder) * 2) + Remainder] = MatrixResult[0];
            SharedMemMatrix[(1 * QbitPower) + ((Aposition - Remainder) * 2) + Remainder] = MatrixResult[1];
            SharedMemMatrix[(0 * QbitPower) + ((Aposition + 1 - Remainder1) * 2) + Remainder1] = MatrixResult[2];
            SharedMemMatrix[(1 * QbitPower) + ((Aposition + 1 - Remainder1) * 2) + Remainder1] = MatrixResult[3];
           
            C[(Bposition * 64) + (0 * QbitPower) + ((Aposition - Remainder) * 2) + Remainder] = MatrixResult[0];
            C[(Bposition * 64) + (1 * QbitPower) + ((Aposition - Remainder) * 2) + Remainder] = MatrixResult[1];
            C[(Bposition * 64) + (0 * QbitPower) + ((Aposition + 1 - Remainder1) * 2) + Remainder1] = MatrixResult[2];
            C[(Bposition * 64) + (1 * QbitPower) + ((Aposition + 1 - Remainder1) * 2) + Remainder1] = MatrixResult[3];
            
            __syncthreads();
        } 
    }
    __syncthreads();
}

uint64_t QbitPosition[6];

uint64_t QuBitIndexCalculator(uint64_t IndexCounter)
{
    bool BinaryIndexCounter[6];
    uint64_t Result = 0;
    for (uint64_t counter = 0; counter < 6; counter++)
    {
        uint64_t QbitVariable = pow(2, QbitPosition[counter]);
        BinaryIndexCounter[counter] = IndexCounter & (1 << counter);
        if (BinaryIndexCounter[counter])
            Result += pow(2, QbitPosition[counter]);
    }
    // cout << "QuBitIndexCalculator" << Result << '\t' << IndexCounter << '\t' << BinaryIndexCounter[0] << '\t' << BinaryIndexCounter[1] << '\t' << BinaryIndexCounter[2] << '\t' << BinaryIndexCounter[3] << '\t' << BinaryIndexCounter[4] << '\t' << BinaryIndexCounter[5] << '\n';
    return Result;
}

int main(int ArgumentCount, char **ArgumentValue)
{
    FILE *TraceFilePointer;
    char *TraceFileName;
    float *Umatrix[6];
    float Filedata = 0.0;
    float *InputMatrixfromFile;
    float *InputSortedMatrixIndex;
    float *InputSortedMatrix;
    float *OutputMatrix;

    if (!(ArgumentCount == 2))
    {
        cout << "Error: Enter Input File" << ArgumentCount - 1 << '\n';
        exit(EXIT_FAILURE);
    }
    // cout << "A" << '\n';
    TraceFileName = ArgumentValue[1];
    TraceFilePointer = fopen(TraceFileName, "r");
    if (TraceFilePointer == NULL)
    {
        cout << "Error: Unable to open file " << TraceFileName << '\n';
        exit(EXIT_FAILURE);
    }
    InputMatrixfromFile = (float *)malloc(pow(2, 30) * sizeof(float));

    for (uint64_t RowCount = 0; RowCount < 6; RowCount++)
        Umatrix[RowCount] = (float *)malloc(4 * sizeof(float));

    // cout << "AB" << '\n';

    for (uint64_t RowCount = 0; RowCount < 6; RowCount++)
    {
        fscanf(TraceFilePointer, "%f %f", &Umatrix[RowCount][0], &Umatrix[RowCount][1]); // Line1
        fscanf(TraceFilePointer, "%f %f", &Umatrix[RowCount][2], &Umatrix[RowCount][3]); // Line2
    }

    while ((fscanf(TraceFilePointer, "%f", &Filedata)) == 1)
    {
        InputMatrixfromFile[InputMatrixSize] = Filedata;
        InputMatrixSize += 1;
    }
    fclose(TraceFilePointer);

    for (uint64_t RowCount = 0; RowCount < 6; RowCount++)
    {
        QbitPosition[5 - RowCount] = InputMatrixfromFile[InputMatrixSize - 1];
        InputMatrixSize = (InputMatrixSize - 1);
    }

    InputSortedMatrixIndex = (float *)malloc((InputMatrixSize) * sizeof(float));
    InputSortedMatrix = (float *)malloc((InputMatrixSize) * sizeof(float));
    OutputMatrix = (float *)malloc((InputMatrixSize) * sizeof(float));

    hipError_t err = hipSuccess;
    size_t InputMatrixFloatSize = InputMatrixSize * sizeof(float);
    size_t QuantumGateFloatSize = 6 * 4 * sizeof(float);
    // Allocate the host input vector A
    float *h_A = (float *)malloc(QuantumGateFloatSize);
    // Allocate the host input vector B
    float *h_B = (float *)malloc(InputMatrixFloatSize);
    // Allocate the host input vector C
    float *h_C = (float *)malloc(InputMatrixFloatSize);

    if ((h_A == NULL) || (h_B == NULL) || (h_C == NULL))
        cout << "Memory Allocate failed for host vectors" << '\n';

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, QuantumGateFloatSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, InputMatrixFloatSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, InputMatrixFloatSize);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /////////////////////////////////////////////////////Input Reordering///////////////////////////////////////////////////////
    for (uint64_t Index = 0; Index < InputMatrixSize; Index += pow(2, 6))
    {
        if (Index == 0)
        {
            for (uint64_t IndexIndex = 0; IndexIndex < pow(2, 6); IndexIndex++)
                InputSortedMatrixIndex[IndexIndex] = QuBitIndexCalculator(IndexIndex);
        }
        else
        {
            for (uint64_t IndexIndex = Index; IndexIndex < (Index + pow(2, 6)); IndexIndex++)
            {
                uint64_t IndexVariable = IndexIndex - pow(2, 6);
                InputSortedMatrixIndex[IndexIndex] = InputSortedMatrixIndex[IndexVariable] + 1;
            }

        CheckDuplicate:
            bool MatchFlag = false;
            for (uint64_t Search = 0; Search < Index; Search++)
            {
                if ((InputSortedMatrixIndex[Index]) == (InputSortedMatrixIndex[Search]))
                {
                    MatchFlag = true;
                    break;
                }
            }
            if (MatchFlag)
            {
                for (uint64_t IndexIndex = Index; IndexIndex < (Index + pow(2, 6)); IndexIndex++)
                    InputSortedMatrixIndex[IndexIndex] += 1;
                goto CheckDuplicate;
            }
        }
    }

    ////////////////////////////////////////////

    for (uint64_t QuantGateCounter = 0; QuantGateCounter < 6; QuantGateCounter++)
    {
        for (uint64_t Bsearch = 0; Bsearch < 4; Bsearch++)
        {
            h_A[(QuantGateCounter * 4) + Bsearch] = Umatrix[QuantGateCounter][Bsearch];
            // cout << "h_A[" << ((QuantGateCounter * 4) + Bsearch) << "]" << '\t' << h_A[(QuantGateCounter * 4) + Bsearch] << '\n';
        }
    }

    for (uint64_t Index = 0; Index < InputMatrixSize; Index++)
    {
        uint64_t ii = InputSortedMatrixIndex[Index];
        InputSortedMatrix[Index] = InputMatrixfromFile[ii];
        h_B[Index] = InputSortedMatrix[Index];
        // cout << "h_B[" << Index << "]" << '\t' << ii << '\t' << h_B[Index] << '\t' << InputSortedMatrix[Index] << '\n';
    }

    // Copy the host input vectors A in host memory to the device input vectors in device memory
    err = hipMemcpy(d_A, h_A, QuantumGateFloatSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors B in host memory to the device input vectors in device memory
    err = hipMemcpy(d_B, h_B, InputMatrixFloatSize, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // cout<<"Input Matrix Size:"<<InputMatrixSize<<'\n'<<'\n';

    dim3 BlockCount(InputMatrixSize / (32 * 2), 1);
    dim3 ThreadCount(16, 1);

    struct timeval begin, end;
    gettimeofday(&begin, NULL);
    QuantumGate<<<BlockCount, ThreadCount>>>(d_A, d_B, d_C, 32, InputMatrixSize / (32 * 2));
    gettimeofday(&end, NULL);
    uint64_t time_in_us = 1e6 * (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec);
    // cout << "Run Time -> " << time_in_us << " us" << '\n';
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch QuantumGate kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    // printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, InputMatrixFloatSize, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    ///////////////////////////////////////////////////////////////////////////

    // for (uint64_t i = 0; i < InputMatrixSize; i++)
    // printf("%.3f\n", h_C[i]);
    // cout << "########################################" << '\n';

    for (uint64_t i = 0; i < (InputMatrixSize); i++)
    {
        uint64_t TempIndex = InputSortedMatrixIndex[i];
        OutputMatrix[TempIndex] = h_C[i];
    }

    ///////////////////////////////////////////////////////////////////////////
    // Free device global memory
    err = hipFree(d_A);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for (uint64_t i = 0; i < InputMatrixSize; i++)
        printf("%.3f\n", OutputMatrix[i]);

    return 0;
}
